
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 10

__global__ void add(int *dev_a, int *dev_b, int *dev_c)
{
    int tid = threadIdx.x;
    if(tid < N)
    {
        dev_c[tid] = dev_a[tid] + dev_b[tid];
    }
}

int main()
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, sizeof(int) * N);
    hipMalloc((void**)&dev_b, sizeof(int) * N);
    hipMalloc((void**)&dev_c, sizeof(int) * N);

    for(int i = 0;i < N;i++)
    {
        a[i] = i;
        b[i] = i*i;
    }

    hipMemcpy(dev_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

    add<<<1, N>>>(dev_a, dev_b ,dev_c);

    hipMemcpy(c, dev_c, sizeof(int)*N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}