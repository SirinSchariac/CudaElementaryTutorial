
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void AddKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int len = 5;
    const int a[len] = {1, 2, 3, 4, 5};
    const int b[len] = {11, 22, 33, 44, 55};

    int c[len] = {0};

    int *dev_a, *dev_b, *dev_c;

    //allocate memory on GPU
    hipMalloc((void**)&dev_a, len * sizeof(int));
    hipMalloc((void**)&dev_b, len * sizeof(int));
    hipMalloc((void**)&dev_c, len * sizeof(int));

    hipMemcpy(dev_a, a, len*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, len*sizeof(int), hipMemcpyHostToDevice);

    AddKernel<<<1, len>>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, len*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for(int i = 0;i < len; i++)
    {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}